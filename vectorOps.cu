#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "vectorOps.h"
#include "utils.h"



__global__ void vecAddKernel(float* A, float* B, float* C, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    if( i < n )
    {
        C[i] = A[i] + B[i];
    }

}

void vecAdd(float* A, float* B, float* C, int n)
{
    unsigned int THREADS_PER_BLOCK = 256;
    
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;
    
    CUDA_CHECK_RETURN(hipMalloc( (void **) &d_A, size));
    CUDA_CHECK_RETURN(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    
    CUDA_CHECK_RETURN(hipMalloc( (void **) &d_B, size));
    CUDA_CHECK_RETURN(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
    
    CUDA_CHECK_RETURN(hipMalloc( (void **) &d_C, size));
    
    vecAddKernel<<<ceil(n/(float) THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(d_A, d_B, d_C,n);
    
    CUDA_CHECK_RETURN(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
    
    CUDA_CHECK_RETURN(hipFree(d_A));
    CUDA_CHECK_RETURN(hipFree(d_B));
    CUDA_CHECK_RETURN(hipFree(d_C));
    
}

